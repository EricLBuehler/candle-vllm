
#include <hip/hip_runtime.h>
#include <stdint.h>

#ifndef USE_ROCM
  #define VLLM_LDG(arg) __ldg(arg)
#else
  #define VLLM_LDG(arg) *(arg)
#endif

template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

// Without neox
extern "C" __global__ void rotary_embedding_kernel_u8(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  uint8_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  uint8_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const uint8_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const uint8_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const uint8_t* cos_ptr = cache_ptr;
  const uint8_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint8_t, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint8_t, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_u32(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  uint32_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  uint32_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const uint32_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const uint32_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const uint32_t* cos_ptr = cache_ptr;
  const uint32_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint32_t, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint32_t, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_i64(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  int64_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  int64_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const int64_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const int64_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const int64_t* cos_ptr = cache_ptr;
  const int64_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int64_t, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int64_t, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_f32(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  float* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  float* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const float* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const float* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const float* cos_ptr = cache_ptr;
  const float* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<float, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<float, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_f64(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  double* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  double* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const double* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const double* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const double* cos_ptr = cache_ptr;
  const double* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<double, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<double, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_f16(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  int16_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  int16_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const int16_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const int16_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const int16_t* cos_ptr = cache_ptr;
  const int16_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_bf16(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  int16_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  int16_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const int16_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const int16_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const int16_t* cos_ptr = cache_ptr;
  const int16_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, false>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, false>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

// With neox
extern "C" __global__ void rotary_embedding_kernel_u8_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  uint8_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  uint8_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const uint8_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const uint8_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const uint8_t* cos_ptr = cache_ptr;
  const uint8_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint8_t, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint8_t, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_u32_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  uint32_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  uint32_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const uint32_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const uint32_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const uint32_t* cos_ptr = cache_ptr;
  const uint32_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint32_t, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<uint32_t, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_i64_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  int64_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  int64_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const int64_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const int64_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const int64_t* cos_ptr = cache_ptr;
  const int64_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int64_t, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int64_t, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_f32_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  float* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  float* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const float* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const float* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const float* cos_ptr = cache_ptr;
  const float* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<float, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<float, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_f64_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  double* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  double* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const double* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const double* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const double* cos_ptr = cache_ptr;
  const double* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<double, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<double, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_f16_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  int16_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  int16_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const int16_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const int16_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const int16_t* cos_ptr = cache_ptr;
  const int16_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

extern "C" __global__ void rotary_embedding_kernel_bf16_neox(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  int16_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  int16_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const int16_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const int16_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const int16_t* cos_ptr = cache_ptr;
  const int16_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, true>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<int16_t, true>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}
