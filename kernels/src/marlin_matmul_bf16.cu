#include "hip/hip_runtime.h"
#include "marlin_gptq_cuda_kernel.cuh"
extern "C" void marlin_4bit_bf16(const void* A, const void* B, void* scales, void* zeros, void* g_idx, void* C, int prob_m, int prob_k, 
                 int prob_n, void* workspace, int groupsize, int64_t stream) {
    marlin_matmul<hip_bfloat16>(A, B, scales, C, prob_m, prob_k, prob_n, workspace, groupsize, stream);
}